
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void firstParallel()
{
  printf("This is running in parallel.\n");
}

int main()
{
  firstParallel<<<2, 3>>>();
  hipDeviceSynchronize();
}
