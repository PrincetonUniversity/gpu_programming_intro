
#include <hip/hip_runtime.h>
#include <stdio.h>

void CPUFunction() {
  printf("Hello world from the CPU.\n");
}

__global__ void GPUFunction() {
  printf("Hello world from the GPU.\n");
}

int main() {
  // function to run on the cpu
  CPUFunction();

  // function to run on the gpu
  GPUFunction<<<1, 1>>>();
  
  // kernel execution is asynchronous so sync on its completion
  hipDeviceSynchronize();
}
